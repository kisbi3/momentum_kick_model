
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <thread>
#include <iomanip>


__device__ double lightcone(double pti, double yi, double sqrSnn, double mp, double m){
    // printf("%f\t%f\n", pti, yi);
    // printf("2");

    // double yi = rapidityinit(pti);
    double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mpi
    // double yb = __device__ double acosh(sqrSnn/(2.*mp));
    // printf("3\n");

    double squareroot=sqrt(m*m+pti*pti);
    // printf("%f\t%f\n", pti, yi);
    // printf("4\n");
    // double yiabs = abs(yi);

    // printf("5\n");
    // printf("%f, %f\n", pti, yi);
    double result = (squareroot/m)*exp(yi-yb);
    // printf("6\n");


    // printf("%f\t%f\n", yi, yiabs);
    // printf("%f, %f, %f\n", pti, yi, result);

    // std::cout<<pti<<std::setw(8)<<yi<<std::setw(15)<<exp(yiabs-yb)<<std::setw(15)<<squareroot/mb<<std::endl;
    // return (squareroot/m)*exp(yi-yb);
    // return (squareroot/m)*exp(fabs(yi)-yb);
    return result;

}

__device__ double integralAridge(double pti, double yi){
    // printf("%f\t%f\n", pti, yi);
    double sqrSnn = 200.;
    double mp = 0.938272046;
    double a = 0.5;
    double T = 0.5;
    double md = 1.;
    double m = 0.13957018;

    // printf("%f\t%f\n", pti, yi);
    // printf("1\n");
    double x = lightcone(pti, yi, sqrSnn, mp, m);
    // printf("%f, %f, %f\n", pti, yi, x);

    // double yi = rapidityinit(pti);
    // double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mpi
    // double squareroot=sqrt(m*m+pti*pti);
    // double yiabs = std::fabs(yi);
    // // std::cout<<pti<<std::setw(8)<<yi<<std::setw(15)<<exp(yiabs-yb)<<std::setw(15)<<squareroot/mb<<std::endl;
    // double x = (squareroot/m)*exp(yiabs-yb);


    // double squareroot=sqrt(m*m+pti*pti);
    // printf("%f\t%f\n", pti, yi);
    if(x>=1.){
        return 0.;
        // cal[index] = 0.;
    }
    else{
        return pti*pow(1-x,a)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
        // cal[index] = pti*pow(1-x,a)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
    }
}

// cal의 x를 pti, y를 yi로 놓고 계산하자. -> x : block / y : thread
__global__ void integrate(int n, double pti_start, double pti_end, double *pti, double yi_start, double yi_end, double *yi, double *sum){
    // printf("%d\n",blockIdx.x);
    // printf("%f\t%f\n", yi_end, yi_start);
    double dyi = ((yi_end-yi_start)/gridDim.x)*100;
    double dpti = (pti_end-pti_start)/blockDim.x;
    pti[blockIdx.x] = dpti*blockIdx.x+pti_start;
    yi[threadIdx.x] = dyi*threadIdx.x+yi_start;
    // pti = dpti*blockIdx.x+pti_start;
    // yi = dyi*threadIdx.x+yi_start;
    // printf("%d\n",blockIdx.x);
    // printf("%f\t%f\n", pti, yi);
    sum[blockIdx.x*blockDim.x+threadIdx.x] = integralAridge(pti[blockIdx.x], yi[threadIdx.x])*dyi*dpti;
    // printf("%d\n",blockIdx.x);
    // int index = blockIdx.x*blockDim.x+threadIdx.x;
    // cal[index] = integralAridge(cal[blockIdx.x], cal[blockIdx.y])
    // sum[index] = cal[index]*dx*dy;
}

// __global__ void integrate(double *pti, double dpti, double *yi, double dyi, double *sum){
//     // sum[blockIdx.x*gridDim.x+threadIdx.x] = integralAridge(pti[blockIdx.x], yi[threadIdx.x])*dyi*dpti;
//     sum[blockIdx.x+threadIdx.x*gridDim.x] = integralAridge(pti[blockIdx.x], yi[threadIdx.x])*dyi*dpti;
//     // printf("%f\t%f\n", pti[blockIdx.x], yi[threadIdx.x]);
// }

int main()
{
    clock_t start, finish;
    double duration;
    start = clock();
    using std::cout;
    using std::endl;
    using std::setw;

    // double dyi, dphii, sum, totalsum, phii, yi, dpti, pti, sum2, resultsum;
    double pti_start, pti_end, yi_start, yi_end, totalsum, Aridge;
    // double dyi, dpti;
    // int i, j, k, nyi, npti, nphii, check2;
    int nyi, npti;

    nyi = 100000;
    npti = 100000;

    pti_start = 0.;
    pti_end = 10.;
    yi_start = 0.;
    yi_end = 10.;

    // dyi = double((0.0+10.)/nyi);
    // dpti = double((0.+10.)/npti);

    // dyi = double ((yi_end-yi_start)/nyi);
    // dpti = double ((pti_end-pti_start)/npti);

    // dphii = double (M_PI+M_PI)/nphii;

    double *pti, *yi, *sum;
    // double *sum;

    // double *yi_cpu = (double*)malloc((nyi/100)*sizeof(double));
    // double *pti_cpu = (double*)malloc(npti*sizeof(double));
    double *sum_cpu = (double*)malloc(npti*(nyi/100)*sizeof(double));

    hipMalloc((void**) &pti, npti*sizeof(double));
    hipMalloc((void**) &yi, (nyi/100)*sizeof(double));
    hipMalloc((void**) &sum, npti*(nyi/100)*sizeof(double));

    // double sum_cpu = (double*)malloc(npti*nyi*sizeof(double));


    // sum = 0.;
    // pti = 0.0;  //적분을 pt=0부터 하는것이 옳은가? 원통좌표계에서의 적분인데?
    // yi = 0.;    //0~4 적분한 후 x2할 것.

    // for(int i = 0;i<npti;i++){
    //     pti_cpu[i] = ((10.-0.)/npti)*i+pti_start;
    // }
    // cudaMemcpy(pti,pti_cpu, npti*sizeof(double), cudaMemcpyHostToDevice);

    totalsum = 0.;
    yi_end = 0.1;
    for(int k=0;k<100;k++){
        // for(int j=0;j<int(nyi/100);j++){
        //     yi_cpu[j] = ((10.-0.)/nyi)*j+yi_start;
        // }
        // cudaMemcpy(yi, yi_cpu, (nyi/100)*sizeof(double), cudaMemcpyHostToDevice);
        // cout<<"1"<<endl;
        // integrate<<<npti,int(nyi/100)>>>(pti, (pti_end-pti_start)/npti, yi, (yi_end-yi_start)/nyi, sum);
        printf("%f\t%f\n", yi_end, yi_start);
        integrate<<<npti,int(nyi/100)>>>(k, pti_start, pti_end, pti, yi_start, yi_end, yi, sum);
        printf("%f\t%f\n", yi_end, yi_start);
        // cout<<"222222"<<endl;
        hipMemcpy(sum_cpu, sum, npti*(nyi/100)*sizeof(double), hipMemcpyDeviceToHost);
        // cout<<"1"<<endl;
        for(int i=0; i<npti*(nyi/100);i++){
            totalsum += sum_cpu[i];
        }   //이 루프가 가장 느린듯
        printf("%f\t%f\t%f\n\n", yi_end, yi_start, totalsum);
        // cout<<"2"<<endl;
        yi_start += 0.1;
        yi_end += 0.1;
    }

    hipFree(pti);
    hipFree(yi);
    hipFree(sum);

    // free(pti_cpu);
    // free(yi_cpu);
    free(sum_cpu);

    totalsum *= 4.*M_PI;
    Aridge = 1/totalsum;
    cout<<totalsum<<setw(20)<<Aridge<<endl;
    finish = clock();
    duration = (double)(finish-start)/CLOCKS_PER_SEC;
    cout<<"time : "<<duration<<" sec"<<endl;

    return 0;
}





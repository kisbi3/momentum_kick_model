
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <thread>
#include <time.h>

// 11/11 cuda 연습 - 여기까지.



// __global__ void function(double *x, int i, double dist){
//     dist = 3*x[i]*x[i];
// }

// double function(){

// }

//gpu를 이용하기 위해서는 '__global__'실행하려는 함수 앞에 붙여야 함.
__global__ void saxpy(int n, double *x, double *y, double dx, double dy, double *sum){
    // int i = blockIdx.x*blockDim.x;

    // printf("%d", blockIdx.x);


    int i = blockIdx.x;
    int j = threadIdx.x;
    // double *func;


    // printf("%d\n",j);

    int index = blockIdx.x*n+threadIdx.x;


    // __shared__ void function(double *x, double *y, double *func){
    //     *func = x*x+y*y;
    // }
    // double dx = (b-a)/n;
    sum[index] = (x[i]*x[i]+y[j]*y[j])*dx*dy;

    // func = function(&x[i], &y[j], &func);
    // sum[index] = dist*dx*dy;

    // double x_temp = x[i];
    // double y_temp = y[j];
    // sum[index] = x_temp*y_temp;
    // sum[index] = x[i]*dx+y[j]*dy;


    // printf("%f\t%f\t%f\t%d\n",x[i],y[j],sum[index],index);


    // sum[index] = x[i]*dx;
    
    // double dist = function(x, i, dist);
    // sum[i] = dist*dx;

    // if( i < n )
    //     sum[i] = a*x[i]+b;
}

double totalintegral = 0.;

void func(int n, double a_1, double b_1, double a_2, double b_2){
    // double a_1 = 0., b_1 = 10.;
    // double a_2 = 0., b_2 = 10.;
    // double a_3 = 0., b_3 = 5.;

    double dx, dy;

    double *h_x = (double*)malloc(n*(sizeof(double)));
    double *h_y = (double*)malloc(n*(sizeof(double)));
    double *h_sum = (double*)malloc(n*n*(sizeof(double)));

    // double x[10000] = {0.}, sum;

    double *x;
    double *y;
    double *sum;

    hipMalloc((void**) &x, n*(sizeof(double)));
    hipMalloc((void**) &y, n*(sizeof(double)));
    hipMalloc((void**) &sum, n*n*(sizeof(double)));

    // std::cout<<'1'<<std::endl;
    dx = (b_1-a_1)/double(n);
    dy = (b_2-a_2)/double(n);
    // for(int i = 0; i<n; i++){
    //     for(int j = 0; j<n; j++){
    //         h_x[i][j] = a_1+a_2 + ((b_1-a_1)/n)*i+((b_2-a_2)/n)*j;
    //     } 
    //     // std::cout<<h_x[i]<<std::endl;
    // }

    // std::cout<<dx<<std::setw(20)<<dy<<std::endl;

    for (int i = 0; i<n; i++){
        h_x[i] = a_1 + ((b_1-a_1)/n)*i;
        h_y[i] = a_2 + ((b_2-a_2)/n)*i;

        // std::cout<<h_x[i]<<std::setw(20)<<h_y[i]<<std::endl;
    }
    
    // std::cout<<std::endl<<std::endl;

    // std::cout<<'1'<<std::endl;

    hipMemcpy(x, h_x, n*(sizeof(double)), hipMemcpyHostToDevice);
    hipMemcpy(y, h_y, n*(sizeof(double)), hipMemcpyHostToDevice);
    hipMemcpy(sum, h_sum, n*n*(sizeof(double)), hipMemcpyHostToDevice);

    saxpy<<<n,n>>>(n, x, y, dx, dy, sum);

    hipMemcpy(h_sum, sum, n*n*(sizeof(double)), hipMemcpyDeviceToHost);

    double total = 0.;
    for(int i = 0; i<n; i++){
        for(int j = 0; j<n; j++){
            double k = h_sum[i+n*j];
            total += k;
            // std::cout<<h_sum[i]<<std::endl;            
        }
        
    }

    totalintegral += total;

    hipFree(x);
    hipFree(sum);

    free(h_x);
    free(h_sum);
}

int main(void)
{
    // using std::thread;
    // using std::cout;
    // using std::endl;
    // using std::setw;
    time_t start, end;

    start = time(NULL);

    // std::cout<<'1'<<std::endl;
    int n = 10000;

    // double a_1 = 0., b_1 = 10.;
    // double a_2 = 0., b_2 = 10.;

    std::thread t1(func, n, 0., 5., 0., 5.);
    std::thread t2(func, n, 0., 5., 5., 10.);
    std::thread t3(func, n, 5., 10., 0., 5.);
    std::thread t4(func, n, 5., 10., 5., 10.);

    t1.join();
    t2.join();
    t3.join();
    t4.join();
    // int size = n*n*(sizeof(double));

    // double dx, dy;
    // // double *x, *sum;

    // double *h_x = (double*)malloc(n*(sizeof(double)));
    // double *h_y = (double*)malloc(n*(sizeof(double)));
    // double *h_sum = (double*)malloc(n*n*(sizeof(double)));

    // // double x[10000] = {0.}, sum;

    // double *x;
    // double *y;
    // double *sum;

    // cudaMalloc((void**) &x, n*(sizeof(double)));
    // cudaMalloc((void**) &y, n*(sizeof(double)));
    // cudaMalloc((void**) &sum, n*n*(sizeof(double)));

    // double a_1 = 0., b_1 = 10.;
    // double a_2 = 0., b_2 = 10.;
    // // double a_3 = 0., b_3 = 5.;

    // // std::cout<<'1'<<std::endl;
    // dx = (b_1-a_1)/double(n);
    // dy = (b_2-a_2)/double(n);
    // // for(int i = 0; i<n; i++){
    // //     for(int j = 0; j<n; j++){
    // //         h_x[i][j] = a_1+a_2 + ((b_1-a_1)/n)*i+((b_2-a_2)/n)*j;
    // //     } 
    // //     // std::cout<<h_x[i]<<std::endl;
    // // }

    // // std::cout<<dx<<std::setw(20)<<dy<<std::endl;

    // for (int i = 0; i<n; i++){
    //     h_x[i] = a_1 + ((b_1-a_1)/n)*i;
    //     h_y[i] = a_2 + ((b_2-a_2)/n)*i;

    //     // std::cout<<h_x[i]<<std::setw(20)<<h_y[i]<<std::endl;
    // }
    
    // // std::cout<<std::endl<<std::endl;

    // // std::cout<<'1'<<std::endl;

    // cudaMemcpy(x, h_x, n*(sizeof(double)), cudaMemcpyHostToDevice);
    // cudaMemcpy(y, h_y, n*(sizeof(double)), cudaMemcpyHostToDevice);
    // cudaMemcpy(sum, h_sum, n*n*(sizeof(double)), cudaMemcpyHostToDevice);

    // // std::cout<<'1'<<std::endl;



    // // saxpy<<<n,n>>>(n, x, y, dx, dy, sum);

    // // std::cout<<'1'<<std::endl;
    // // 'saxpy'의 함수를 nx3개의 gpu thread가 실행한다는 의미
    // // saxpy<<<b, n>>>();
    // // 에서 saxpy는 함수의 이름, b는 함수를 수행할 block의 개수, n은 다시 하나의 thread block 안에 몇 개의 thread가 존재하는지를 정하는 것.

    // cudaMemcpy(h_sum, sum, n*n*(sizeof(double)), cudaMemcpyDeviceToHost);

    // total = 0.;
    // for(int i = 0; i<n; i++){
    //     for(int j = 0; j<n; j++){
    //         double k = h_sum[i+n*j];
    //         total += k;
    //         // std::cout<<h_sum[i]<<std::endl;            
    //     }
        
    // }

    // cudaFree(x);
    // cudaFree(sum);

    // free(h_x);
    // free(h_sum);

    // std::cout<<total<<std::endl;
    std::cout<<totalintegral<<std::endl;

    end = time(NULL);
    // std::cout<<double(end-start)<<std::endl;
    std::cout<<"걸린 시간 : "<<double(end-start)<<std::endl;

    return 0;
}
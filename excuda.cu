// #include <stdio.h>

// __global__
// void saxpy(int n, float a, float *x, float *y)
// {
//   int i = blockIdx.x*blockDim.x + threadIdx.x;
//   if (i < n) y[i] = a*x[i] + y[i];
// }

// int main(void)
// {
//   int N = 1<<20;
//   float *x, *y, *d_x, *d_y;
//   x = (float*)malloc(N*sizeof(float));
//   y = (float*)malloc(N*sizeof(float));

//   cudaMalloc(&d_x, N*sizeof(float)); 
//   cudaMalloc(&d_y, N*sizeof(float));

//   for (int i = 0; i < N; i++) {
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//   }

//   cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
//   cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

//   // Perform SAXPY on 1M elements
//   saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

//   cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = max(maxError, abs(y[i]-4.0f));
//   printf("Max error: %f\n", maxError);

//   cudaFree(d_x);
//   cudaFree(d_y);
//   free(x);
//   free(y);
// }


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void saxpy(int n, float a, float *__restrict__ x, float *__restrict__ y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < n )
    y[i] = a * x[i] + y[i];
}

int main()
{
  std::cout<<'1'<<std::endl;
  int N = 1 << 16;
  
  int size = N * sizeof(float);
  float *h_x = (float*)malloc(size);
  float *h_y = (float*)malloc(size);
  
  float *d_x;
  float *d_y;
  
  hipMalloc((void**) &d_x, size);
  hipMalloc((void**) &d_y, size);
  
  std::cout<<'1'<<std::endl;
  for(int i=0; i < N; i++)
  {
    h_x[i] = 2.0;
    h_y[i] = 2.0;
  }
  
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);
  
  std::cout<<'1'<<std::endl;
  saxpy<<<256, 256>>>(N, 2.0, d_x, d_y);
  
  hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

  std::cout<<*h_y<<std::endl;
  
  hipFree(d_x);
  hipFree(d_y);
  
  free(h_x);
  free(h_y);

  
  
  return 0;
}
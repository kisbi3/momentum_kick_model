// #define NSTEP 1000
#define NKERNEL 20


int NSTEP = 1000;
// start CPU wallclock timer
for(int istep=0; istep<NSTEP; istep++){
    for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
      shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
      hipStreamSynchronize(stream);
    }
  }
  //end CPU wallclock time


// start wallclock timer
for(int istep=0; istep<NSTEP; istep++){
    for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
      shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
    }
    hipStreamSynchronize(stream);
  }
//end wallclock timer


bool graphCreated=false;
hipGraph_t graph;
hipGraphExec_t instance;
for(int istep=0; istep<NSTEP; istep++){
  if(!graphCreated){
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
      shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
    }
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
    graphCreated=true;
  }
  hipGraphLaunch(instance, stream);
  hipStreamSynchronize(stream);
}
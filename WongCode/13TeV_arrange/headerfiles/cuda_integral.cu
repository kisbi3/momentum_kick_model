#include "hip/hip_runtime.h"
// #include "integral.hpp"
#include "cuda_function.cuh"
#include "cuda_integral.cuh"

// static const int blockSize = 1024;

// __device__ double inside_cuda_function(double (*func)(double, double), double x, double y){
//     return x*x+y*y;
// }

__global__ void integrate(double x_st, double x_end, double *x, double y_st, double y_end, double *y, double *sum){
    double dx = (x_end-x_st)/gridDim.x;
    double dy = (y_end-y_st)/blockDim.x;

    sum[blockIdx.x*blockDim.x+threadIdx.x] = main_function(dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    // __syncthreads();
}

__global__ void sumCommMultiBlock(double *gArr, int arraySize, double *gOut) {
    static const int blockSize = 1024;
    
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockDim.x;
    const int gridSize = blockDim.x*gridDim.x;
    double sum = 0;
    // printf("%f\n", gArr[threadIdx.x]);
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockDim.x/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}



extern "C" double cuda_twodimsumarray(double *gArr, int bin_x, int bin_y, double *gOut){
    double totaldist = 0.;
    sumCommMultiBlock<<<bin_x,1024>>>(gArr, bin_x*bin_y, gOut);
    sumCommMultiBlock<<<1,1024>>>(gOut, bin_x, gOut);
    hipDeviceSynchronize();
    hipMemcpy(&totaldist, gOut, sizeof(double), hipMemcpyDeviceToHost);
    return totaldist;
}

// double cuda_function(double (*func)(double, double), double x, double y){

    
//     return func(x, y);
// }

extern "C" double cuda_secondintegral(double x_st, double x_end, int bin_x, double y_st, double y_end, int bin_y){

    double totalsum = 0.;
    double *x, *y, *sum, *sumdist;
    hipMalloc((void**) &x, bin_x*sizeof(double));
    hipMalloc((void**) &sumdist, bin_x*sizeof(double));

    // if(bin_y>1000){
    //     #define over
    // }
    
    // #define over (bin_y>1000)
    // // printf("\n%d", bin_y);
    // #ifdef over
    int thread_num = 800;
    if(bin_y>thread_num){
        #define overthread
    }
    // if(bin_y>thread_num){
    // #if bin_y > thread_num
    #ifdef overthread
        // printf("\n1");
        int iteration = bin_y/thread_num;
        double dy = (y_end-y_st)/iteration;
        y_end = dy;
        
        double *sum_cpu = (double*)malloc(bin_x*(bin_y/iteration)*sizeof(double));
        hipMalloc((void**) &y, (bin_y/iteration)*sizeof(double));
        hipMalloc((void**) &sum, bin_x*(bin_y/iteration)*sizeof(double));    
        for(int k=0;k<iteration;k++){
            // double totaldist = 0.;
            // integrate<<<bin_x,1000>>>(func, x_st, x_end, x, y_st, y_end, y, sum);
            integrate<<<bin_x,thread_num>>>(x_st, x_end, x, y_st, y_end, y, sum);
            // sumCommMultiBlock<<<bin_x,1024>>>(sum, bin_x*(bin_y/100), sumdist);
            // sumCommMultiBlock<<<1,1024>>>(sumdist, bin_x, sumdist);
            // hipDeviceSynchronize();
            // hipMemcpy(&totaldist, sumdist, sizeof(double), hipMemcpyDeviceToHost);

            double totaldist = cuda_twodimsumarray(sum, bin_x, bin_y/iteration, sumdist);

            // printf("%f\n", totaldist);
            totalsum += totaldist;
            y_st += dy;
            y_end += dy;
        }
        free(sum_cpu);
    // }
    #else 
    // else{
        // printf("111");
        double *sum_cpu = (double*)malloc(bin_x*bin_y*sizeof(double));
        hipMalloc((void**) &y, bin_y*sizeof(double));
        hipMalloc((void**) &sum, bin_x*bin_y*sizeof(double)); 
        integrate<<<bin_x,bin_y>>>(x_st, x_end, x, y_st, y_end, y, sum);
        totalsum = cuda_twodimsumarray(sum, bin_x, bin_y, sumdist);
        free(sum_cpu);
    // }
    #endif



    //Error Message!!
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipFree(x);
    hipFree(y);
    hipFree(sum);
    hipFree(sumdist);

    // free(sum_cpu);

    return totalsum;
}

// extern "C" double cuda_thirdintegral(double x_st, double x_end, int bin_x, double y_st, double y_end, int bin_y){

//     double totalsum = 0.;
//     double *x, *y, *z, *sum, *sumdist;
//     hipMalloc((void**) &x, bin_x*sizeof(double));
//     hipMalloc((void**) &sumdist, bin_x*sizeof(double));

//     // if(bin_y>1000){
//     //     #define over
//     // }
    
//     // #define over (bin_y>1000)
//     // // printf("\n%d", bin_y);
//     // #ifdef over
//     int thread_num = 800;
//     if(bin_y>thread_num){
//         #define overthread
//     }
//     // if(bin_y>thread_num){
//     // #if bin_y > thread_num
//     #ifdef overthread
//         // printf("\n1");
//         int iteration = bin_y/thread_num;
//         double dy = (y_end-y_st)/iteration;
//         y_end = dy;
        
//         double *sum_cpu = (double*)malloc(bin_x*(bin_y/iteration)*sizeof(double));
//         hipMalloc((void**) &y, (bin_y/iteration)*sizeof(double));
//         hipMalloc((void**) &sum, bin_x*(bin_y/iteration)*sizeof(double));    
//         for(int k=0;k<iteration;k++){
//             // double totaldist = 0.;
//             // integrate<<<bin_x,1000>>>(func, x_st, x_end, x, y_st, y_end, y, sum);
//             integrate<<<bin_x,thread_num>>>(x_st, x_end, x, y_st, y_end, y, sum);
//             // sumCommMultiBlock<<<bin_x,1024>>>(sum, bin_x*(bin_y/100), sumdist);
//             // sumCommMultiBlock<<<1,1024>>>(sumdist, bin_x, sumdist);
//             // hipDeviceSynchronize();
//             // hipMemcpy(&totaldist, sumdist, sizeof(double), hipMemcpyDeviceToHost);

//             double totaldist = cuda_twodimsumarray(sum, bin_x, bin_y/iteration, sumdist);

//             // printf("%f\n", totaldist);
//             totalsum += totaldist;
//             y_st += dy;
//             y_end += dy;
//         }
//         free(sum_cpu);
//     // }
//     #else 
//     // else{
//         // printf("111");
//         double *sum_cpu = (double*)malloc(bin_x*bin_y*sizeof(double));
//         hipMalloc((void**) &y, bin_y*sizeof(double));
//         hipMalloc((void**) &sum, bin_x*bin_y*sizeof(double)); 
//         integrate<<<bin_x,bin_y>>>(x_st, x_end, x, y_st, y_end, y, sum);
//         totalsum = cuda_twodimsumarray(sum, bin_x, bin_y, sumdist);
//         free(sum_cpu);
//     // }
//     #endif



//     //Error Message!!
//     hipError_t err = hipGetLastError();

//     if ( err != hipSuccess )
//     {
//        printf("CUDA Error: %s\n", hipGetErrorString(err));       

//        // Possibly: exit(-1) if program cannot continue....
//     }

//     hipFree(x);
//     hipFree(y);
//     hipFree(sum);
//     hipFree(sumdist);

//     // free(sum_cpu);

//     return totalsum;
// }
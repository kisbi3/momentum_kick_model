#include "hip/hip_runtime.h"
#include "cuda_function.cuh"
#include <cmath>


__device__ double fallof = .5;    //fall off parameter
__device__ double T = .5;    //Temperature, GeV
__device__ double q = .87;    //GeV
__device__ double m = 0.13957018;  //m == mpi
__device__ double mb = 0.13957018; //mb==mpi, GeV
__device__ double md = 1.;   //GeV
__device__ double sqrSnn = 200.;
__device__ double mp = 0.938272046; //Proton mass, GeV


//Jet Parameters
__device__ double Njet=10.;
__device__ double fj=1.;
__device__ double Tjet=0.7;
__device__ double sigmaphizero = .05;
__device__ double ma = 100.;

__device__ double Aridge;

__device__ double main_function(double x, double y){
    double dist = integralAridge(x, y);
    // printf("%f\n", dist);
    return dist;
    // return exp(x)+sqrt(y);
}

__device__ double integralNjet(double pt, double eta, double phi){
    double sigmaphi;
    double constant = Njet/(Tjet*(m+Tjet)*2*M_PI);
    sigmaphi = (sigmaphizero*sigmaphizero*ma*ma)/(ma*ma+pt*pt);
    return (constant/sigmaphi)*exp(((m-sqrt(m*m+pt*pt))/Tjet)-((phi*phi+eta*eta)/(2*sigmaphi)));
}

__device__ double lightcone(double pti, double yi){
    // printf("%f\n", sqrSnn);
    double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mp

    double squareroot=sqrt(m*m+pti*pti);
    // double yiabs = std::fabs(yi);
    return (squareroot/m)*exp(fabs(yi)-yb);
    // return exp(yiabs-yb);
}

__device__ double integralAridge(double pti, double yi){
    double x = lightcone(pti, yi);
    double squareroot=sqrt(m*m+pti*pti);
    // printf("%f\n", x);
    if(x>=1.){
        return 0.;
    }
    else{
        double fall = pow(1-x,0.5);
        printf("%f\n", fall);
        // return pti*pow(1-x,0.5)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
        return pti*sqrt(1.-x)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
        // return pti*exp(x)+yi;
        // return pti*pow(1.-x,a)+yi;
        // double power = pow(1.-x,6.);
        // return sqrt(1.-x);
    }
    
}

// __device__ double lightcone(double pti, double yi){    
//     double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mp

//     double squareroot=sqrt(m*m+pti*pti);
//     double yiabs = std::fabs(yi);
//     return (squareroot/m)*exp(yiabs-yb);
// }

__device__ double RidgeDisi(double pti, double yi){
    double x = lightcone(pti, yi);
    double squareroot=sqrt(m*m+pti*pti);


        if(x>=1.){
            return 0.;
        }
        else{
            return pow(1-x,fallof)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
        }
    
}


__device__ double RidgeDisf(double ptf, double etaf, double phif){
    double etajet = 0.;
    double ptisq = ptf*ptf-2*ptf*q*cos(phif)/cosh(etajet)+q*q/(cosh(etajet)*cosh(etajet));
    double pti;
    if(ptisq<0.0000000001){
        pti = 0.;
    }
    else{
        pti = sqrt(ptisq);
    }
    double E = sqrt(ptf*ptf*cosh(etaf)*cosh(etaf)+m*m);
    double Ei = sqrt(pti*pti+ptf*ptf*sinh(etaf)*sinh(etaf)+m*m);

    double yi = log((Ei+ptf*sinh(etaf))/(Ei-ptf*sinh(etaf)))/2;
    double yf = log((E+ptf*sinh(etaf))/(E-ptf*sinh(etaf)))/2;

    double x = lightcone(pti, yi);

    if (x>=1.){
        return 0.;
    }
    
    else{              
        return (Aridge*RidgeDisi(pti, yi))*sqrt(1.-((mb*mb)/((mb*mb+ptf*ptf)*cosh(yf)*cosh(yf))))*(E/Ei);
    }
    
}
#include "hip/hip_runtime.h"
#include "cuda_function.cuh"
#include <cmath>


__device__ double fallof = 0.5;    //fall off parameter
__device__ double T = 0.5;    //Temperature, GeV
__device__ double q = 0.87;    //GeV
__device__ double m = 0.13957018;  //m == mpi
__device__ double mb = 0.13957018; //mb==mpi, GeV
__device__ double md = 1.0;   //GeV
__device__ double sqrSnn = 200.;
__device__ double mp = 0.938272; //Proton mass, GeV

//fRNk Parameters
__device__ double F1 = .32;
__device__ double F2 = .71;

//Jet Parameters
__device__ double Njet=10.;
__device__ double fj=1.;
__device__ double Tjet=0.7;
__device__ double sigmaphizero = .05;
__device__ double ma = 100.;

__device__ double Aridge;

__device__ double frnk(double pt){
    return F1*exp(F2*pt);
}

__device__ double main_function2(int function_number, double x, double y){
    // double dist = integralAridge(x, y);
    // printf("%f\n", dist);

    if(function_number == 1){
        return integralAridge(x,y);
    }
    else if(function_number == 2 ){
        return x*frnk(x)*RidgeDisf(x,y,1.28);
    }
    else{
        printf("Select function.");
        return 1;
        // exit(1);
    }
    // return func(x, y);
    // return exp(x)+sqrt(y);
}

__device__ double main_function3(int function_number, double x, double y, double z){
    
    //In this function, x == ptf, y == etaf, z == phif
    if(function_number == 2){
        return x*frnk(x)*RidgeDisf(x,y,z);
    }
    else{
        printf("Select function.");
        return 1;
    }
    
}

__device__ double integralNjet(double pt, double eta, double phi){
    double sigmaphi;
    double constant = Njet/(Tjet*(m+Tjet)*2*M_PI);
    sigmaphi = (sigmaphizero*sigmaphizero*ma*ma)/(ma*ma+pt*pt);
    return (constant/sigmaphi)*exp(((m-sqrt(m*m+pt*pt))/Tjet)-((phi*phi+eta*eta)/(2*sigmaphi)));
}

__device__ double lightcone(double pti, double yi){
    // printf("%f\n", sqrSnn);
    double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mp

    double squareroot=sqrt(m*m+pti*pti);
    // double yiabs = std::fabs(yi);
    return (squareroot/m)*exp(fabs(yi)-yb);
    // return exp(yiabs-yb);
}

__device__ double integralAridge(double pti, double yi){
    double x = lightcone(pti, yi);
    // double squareroot=sqrt(m*m+pti*pti);
    if(x>=1.){
        return 0.;
    }
    else{

        return pti*pow(1-x,fallof)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);

    }
    
}


__device__ double RidgeDisi(double pti, double yi){
    double x = lightcone(pti, yi);
    double squareroot=sqrt(m*m+pti*pti);


        if(x>=1.){
            return 0.;
        }
        else{
            return pow(1-x,fallof)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
        }
    
}


__device__ double RidgeDisf(double ptf, double etaf, double phif){
    double etajet = 0.;
    double ptisq = ptf*ptf-2*ptf*q*cos(phif)/cosh(etajet)+q*q/(cosh(etajet)*cosh(etajet));
    double pti;
    if(ptisq<0.0000000001){
        pti = 0.;
    }
    else{
        pti = sqrt(ptisq);
    }
    double E = sqrt(ptf*ptf*cosh(etaf)*cosh(etaf)+m*m);
    double Ei = sqrt(pti*pti+ptf*ptf*sinh(etaf)*sinh(etaf)+m*m);

    double yi = log((Ei+ptf*sinh(etaf))/(Ei-ptf*sinh(etaf)))/2;
    double yf = log((E+ptf*sinh(etaf))/(E-ptf*sinh(etaf)))/2;

    double x = lightcone(pti, yi);

    if (x>=1.){
        return 0.;
    }
    
    else{              
        return (Aridge*RidgeDisi(pti, yi))*sqrt(1.-((mb*mb)/((mb*mb+ptf*ptf)*cosh(yf)*cosh(yf))))*(E/Ei);
    }
    
}
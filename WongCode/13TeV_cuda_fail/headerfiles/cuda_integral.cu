#include "hip/hip_runtime.h"
// #include "integral.hpp"
#include "cuda_function.cuh"
#include "cuda_integral.cuh"

// static const int blockSize = 1024;


__global__ void integrate2(double x_st, double x_end, double *x, double y_st, double y_end, double *y, double *sum, int function_number){
    double dx = (x_end-x_st)/gridDim.x;
    double dy = (y_end-y_st)/blockDim.x;

    sum[blockIdx.x*blockDim.x+threadIdx.x] = main_function2(function_number, dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    // sum[blockIdx.x*blockDim.x+threadIdx.x] = func(dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    // __syncthreads();
}

__global__ void integrate3(double x_st, double x_end, double *x, double y_st, double y_end, double *y, double z_st, double z_end, double *z, double *sum, int function_number){
    double dx = (x_end-x_st)/gridDim.x;
    double dy = (y_end-y_st)/gridDim.y;
    double dz = (z_end-z_st)/blockDim.x;

    sum[blockIdx.x*gridDim.y*gridDim.y+blockIdx.y*blockDim.x+threadIdx.x] = main_function3(function_number, dx*blockIdx.x+x_st, dy*blockIdx.y+y_st, dz*threadIdx.x+z_st)*dx*dy*dz;
    // sum[blockIdx.x*blockDim.x+threadIdx.x] = func(dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    // __syncthreads();
}

__global__ void sumCommMultiBlock(double *gArr, int arraySize, double *gOut) {
    static const int blockSize = 1024;
    
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockDim.x;
    const int gridSize = blockDim.x*gridDim.x;
    double sum = 0;
    // printf("%f\n", gArr[threadIdx.x]);
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockDim.x/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}



extern "C" double cuda_twodimsumarray(double *gArr, int bin_x, int bin_y, double *gOut){
    double totaldist = 0.;
    sumCommMultiBlock<<<bin_x,1024>>>(gArr, bin_x*bin_y, gOut);
    sumCommMultiBlock<<<1,1024>>>(gOut, bin_x, gOut);
    hipDeviceSynchronize();
    hipMemcpy(&totaldist, gOut, sizeof(double), hipMemcpyDeviceToHost);
    return totaldist;
}

// double cuda_function(double (*func)(double, double), double x, double y){

    
//     return func(x, y);
// }

extern "C" double cuda_secondintegral(double x_st, double x_end, int bin_x, double y_st, double y_end, int bin_y, int function_number){

    double totalsum = 0.;
    double *x, *y, *sum, *sumdist;
    hipMalloc((void**) &x, bin_x*sizeof(double));
    hipMalloc((void**) &sumdist, bin_x*sizeof(double));

    // if(function_number == 1){
    //     double *func(x,y) = integralAridge(x,y);
    // }
    // else if(function_number == 2 ){
    //     double *func(x,y) = frnk(x)*RidgeDisf(x,y);
    // }
    // else{
    //     printf("Select function.");
    //     exit();
    // }



    // if(bin_y>1000){
    //     #define over
    // }
    
    // #define over (bin_y>1000)
    // // printf("\n%d", bin_y);
    // #ifdef over
    int thread_num = 800;
    // if(bin_y>thread_num){
    //     #define overthread
    // }
    if(bin_y>thread_num){
    // #if bin_y > thread_num
    // #ifdef overthread
        // printf("\n1");
        int iteration = bin_y/thread_num;
        double dy = (y_end-y_st)/iteration;
        y_end = dy;
        
        double *sum_cpu = (double*)malloc(bin_x*(bin_y/iteration)*sizeof(double));
        hipMalloc((void**) &y, (bin_y/iteration)*sizeof(double));
        hipMalloc((void**) &sum, bin_x*(bin_y/iteration)*sizeof(double));    
        for(int k=0;k<iteration;k++){
            // double totaldist = 0.;
            // integrate<<<bin_x,1000>>>(func, x_st, x_end, x, y_st, y_end, y, sum);
            integrate2<<<bin_x,thread_num>>>(x_st, x_end, x, y_st, y_end, y, sum, function_number);
            // sumCommMultiBlock<<<bin_x,1024>>>(sum, bin_x*(bin_y/100), sumdist);
            // sumCommMultiBlock<<<1,1024>>>(sumdist, bin_x, sumdist);
            // hipDeviceSynchronize();
            // hipMemcpy(&totaldist, sumdist, sizeof(double), hipMemcpyDeviceToHost);

            double totaldist = cuda_twodimsumarray(sum, bin_x, bin_y/iteration, sumdist);

            // printf("%f\n", totaldist);
            totalsum += totaldist;
            y_st += dy;
            y_end += dy;
        }
        free(sum_cpu);
    }
    // #else 
    else{
        // printf("111");
        double *sum_cpu = (double*)malloc(bin_x*bin_y*sizeof(double));
        hipMalloc((void**) &y, bin_y*sizeof(double));
        hipMalloc((void**) &sum, bin_x*bin_y*sizeof(double)); 
        integrate2<<<bin_x,bin_y>>>(x_st, x_end, x, y_st, y_end, y, sum, function_number);
        totalsum = cuda_twodimsumarray(sum, bin_x, bin_y, sumdist);
        free(sum_cpu);
    }
    // #endif



    //Error Message!!
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipFree(x);
    hipFree(y);
    hipFree(sum);
    hipFree(sumdist);

    // free(sum_cpu);

    return totalsum;
}

extern "C" double cuda_thirdintegral(double x_st, double x_end, int bin_x, double y_st, double y_end, int bin_y, double z_st, double z_end, int bin_z, int function_number){

    double totalsum = 0.;
    double *x, *y, *z, *sum, *sumdist;
    hipMalloc((void**) &x, bin_x*sizeof(double));
    hipMalloc((void**) &y, bin_y*sizeof(double));
    hipMalloc((void**) &sumdist, bin_x*sizeof(double));

    int thread_num = 800;


    if(bin_z>thread_num){
    // #if bin_y > thread_num
    // #ifdef overthread
        // printf("\n1");
        int iteration = bin_z/thread_num;
        double dy = (y_end-y_st)/iteration;
        y_end = dy;
        
        double *sum_cpu = (double*)malloc(bin_x*(bin_z/iteration)*sizeof(double));
        hipMalloc((void**) &z, (bin_z/iteration)*sizeof(double));
        hipMalloc((void**) &sum, bin_x*bin_y*(bin_z/iteration)*sizeof(double));    
        for(int k=0;k<iteration;k++){
            // double totaldist = 0.;
            // integrate<<<bin_x,1000>>>(func, x_st, x_end, x, y_st, y_end, y, sum);
            dim3 grid(bin_x, bin_y);
            integrate3<<<grid,thread_num>>>(x_st, x_end, x, y_st, y_end, y, z_st, z_end, z, sum, function_number);

            //sumarray 건드려야 함!!!
            double totaldist = cuda_twodimsumarray(sum, bin_x*bin_y, bin_z/iteration, sumdist);

            // printf("%f\n", totaldist);
            totalsum += totaldist;
            y_st += dy;
            y_end += dy;
        }
        free(sum_cpu);
    }
    // #else 
    else{
        // printf("111");
        double *sum_cpu = (double*)malloc(bin_x*bin_z*sizeof(double));
        hipMalloc((void**) &z, bin_z*sizeof(double));
        hipMalloc((void**) &sum, bin_x*bin_y*bin_z*sizeof(double)); 
        dim3 grid(bin_x, bin_y);
        integrate3<<<grid,bin_z>>>(x_st, x_end, x, y_st, y_end, y, z_st, z_end, z, sum, function_number);
        

        totalsum = cuda_twodimsumarray(sum, bin_x*bin_y, bin_z, sumdist);
        free(sum_cpu);
    }
    // #endif



    //Error Message!!
    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
       printf("CUDA Error: %s\n", hipGetErrorString(err));       

       // Possibly: exit(-1) if program cannot continue....
    }

    hipFree(x);
    hipFree(y);
    hipFree(sum);
    hipFree(sumdist);

    // free(sum_cpu);

    return totalsum;
}
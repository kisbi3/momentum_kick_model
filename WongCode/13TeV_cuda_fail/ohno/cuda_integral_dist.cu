#include "hip/hip_runtime.h"
// #include "integral.hpp"
#include "cuda_function.cuh"
#include "cuda_integral.cuh"

// static const int blockSize = 1024;

// __device__ double inside_cuda_function(double (*func)(double, double), double x, double y){
//     return x*x+y*y;
// }

__device__ double inside_cuda_function(double x, double y){
    double res = x*x+y*y;
    // printf("%f\t%f\t%f\n", x, y, res);
    // return x*x+y*y;
    return res;
}

// __global__ __device__ cuda_function(){

// }

// extern "C" double cuda_func(double x, double y){
    
// }


// __global__ void integrate(double (*func)(double, double),double x_st, double x_end, double *x, double y_st, double y_end, double *y, double *sum){
__global__ void integrate(double x_st, double x_end, double *x, double y_st, double y_end, double *y, double *sum){
    double dx = (x_end-x_st)/gridDim.x;
    double dy = (y_end-y_st)/blockDim.x;

    // printf("%f\t%f\n", dx, dy);
    // printf("%f\t%f\t%f\t%f\t%d\t%d\t%f\t%f\n", x_end, x_st, y_end, y_st, blockDim.x, gridDim.x, dx, dy);
    // if(blockIdx.x==0 && threadIdx.x==0){
    //     printf("%f\t%f\n", y_end, y_st);
    // }
    
    // x[blockIdx.x] = dx*blockIdx.x+x_st;
    // y[threadIdx.x] = dy*threadIdx.x+y_st;
    // sum[blockIdx.x*blockDim.x+threadIdx.x] = inside_cuda_function(x[blockIdx.x], y[threadIdx.x])*dx*dy;
    // sum[blockIdx.x*blockDim.x+threadIdx.x] = inside_cuda_function(dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    // sum[blockIdx.x*blockDim.x+threadIdx.x] =cuda_func(dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    sum[blockIdx.x*blockDim.x+threadIdx.x] = main_function(dx*blockIdx.x+x_st, dy*threadIdx.x+y_st)*dx*dy;
    // if(blockIdx.x==0 && threadIdx.x==0){
    //     printf("%f\n", sum[0]);
    // }
}

__global__ void sumCommMultiBlock(double *gArr, int arraySize, double *gOut) {
    static const int blockSize = 1024;
    
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockDim.x;
    const int gridSize = blockDim.x*gridDim.x;
    double sum = 0;
    // printf("%f\n", gArr[threadIdx.x]);
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockDim.x/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}



extern "C" double cuda_twodimsumarray(double *gArr, int bin_x, int bin_y, double *gOut){
    double totaldist = 0.;
    sumCommMultiBlock<<<bin_x,1024>>>(gArr, bin_x*bin_y, gOut);
    sumCommMultiBlock<<<1,1024>>>(gOut, bin_x, gOut);
    hipDeviceSynchronize();
    hipMemcpy(&totaldist, gOut, sizeof(double), hipMemcpyDeviceToHost);
    return totaldist;
}

// double cuda_function(double (*func)(double, double), double x, double y){

    
//     return func(x, y);
// }

extern "C" double cuda_secondintegral(double x_st, double x_end, int bin_x, double y_st, double y_end, int bin_y){

    double totalsum = 0.;
    int iteration = bin_y/1000;

    double *sum_cpu = (double*)malloc(bin_x*(bin_y/iteration)*sizeof(double));
    double *x, *y, *sum, *sumdist;
    hipMalloc((void**) &x, bin_x*sizeof(double));
    hipMalloc((void**) &y, (bin_y/iteration)*sizeof(double));
    hipMalloc((void**) &sum, bin_x*(bin_y/iteration)*sizeof(double));
    hipMalloc((void**) &sumdist, bin_x*sizeof(double));

    double dy = (y_end-y_st)/iteration;
    y_end = dy;

    for(int k=0;k<iteration;k++){
        // integrate<<<bin_x,1000>>>(func, x_st, x_end, x, y_st, y_end, y, sum);
        integrate<<<bin_x,1000>>>(x_st, x_end, x, y_st, y_end, y, sum);
        // sumCommMultiBlock<<<bin_x,1024>>>(sum, bin_x*(bin_y/100), sumdist);
        // sumCommMultiBlock<<<1,1024>>>(sumdist, bin_x, sumdist);
        // hipDeviceSynchronize();
        // hipMemcpy(&totaldist, sumdist, sizeof(double), hipMemcpyDeviceToHost);
        double totaldist = cuda_twodimsumarray(sum, bin_x, bin_y/iteration, sumdist);
        // printf("%f\n", totaldist);
        totalsum += totaldist;
        y_st += dy;
        y_end += dy;
    }



    hipFree(x);
    hipFree(y);
    hipFree(sum);
    hipFree(sumdist);

    free(sum_cpu);

    return totalsum;
}
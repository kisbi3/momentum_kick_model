
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <thread>
#include <time.h>

// 11/11 cuda 연습 - 여기까지.



// __global__ void function(double *x, int i, double dist){
//     dist = 3*x[i]*x[i];
// }

double function(double x, double y){
    // printf("8\n");
    // printf("%f\n",x*x+y*y);
    // *funcal = x*x+y*y;
    // double *ptr = x*x+y*y;
    return x*x+y*y;
    // return ptr;
}

//gpu를 이용하기 위해서는 '__global__'실행하려는 함수 앞에 붙여야 함.
__global__ void saxpy(int n, double dx, double dy, double *cal, double *sum){
    // int i = blockIdx.x;
    // int j = threadIdx.x;
    // printf("1");
    int index = blockIdx.x*n+threadIdx.x;
    // printf("%f\t%f\n",cal[index],sum[index]);
    sum[index] = cal[index]*dx*dy;
}

// double totalintegral = 0.;

void func(int n, double a_1, double b_1, double a_2, double b_2, double *totalintegral){
    // double a_1 = 0., b_1 = 10.;
    // double a_2 = 0., b_2 = 10.;
    // double a_3 = 0., b_3 = 5.;

    double dx, dy, x, y;

    // double *h_x = (double*)malloc(n*(sizeof(double)));
    // double *h_y = (double*)malloc(n*(sizeof(double)));
    double *h_sum = (double*)malloc(n*n*(sizeof(double)));
    double *h_cal = (double*)malloc(n*n*(sizeof(double)));

    // double x[10000] = {0.}, sum;

    // double *x;
    // double *y;
    double *sum;
    double *cal;

    // cudaMalloc((void**) &x, n*(sizeof(double)));
    // cudaMalloc((void**) &y, n*(sizeof(double)));
    hipMalloc((void**) &sum, n*n*(sizeof(double)));
    hipMalloc((void**) &cal, n*n*(sizeof(double)));

    // std::cout<<'1'<<std::endl;
    dx = (b_1-a_1)/double(n);
    dy = (b_2-a_2)/double(n);
    // for(int i = 0; i<n; i++){
    //     for(int j = 0; j<n; j++){
    //         h_x[i][j] = a_1+a_2 + ((b_1-a_1)/n)*i+((b_2-a_2)/n)*j;
    //     } 
    //     // std::cout<<h_x[i]<<std::endl;
    // }

    // std::cout<<dx<<std::setw(20)<<dy<<std::endl;

    // for (int i = 0; i<n; i++){
    //     h_x[i] = a_1 + ((b_1-a_1)/n)*i;
    //     h_y[i] = a_2 + ((b_2-a_2)/n)*i;

    //     // std::cout<<h_x[i]<<std::setw(20)<<h_y[i]<<std::endl;
    // }
    // printf("1\n");

    // double funcal;
    x = a_1;
    for(int i=0; i<n; i++){
        // printf("%f\n",x);
        y = a_2;
        for(int j=0; j<n; j++){
            // printf("%f\n",y);
            // funcal = function(x,y);
            // double *funcall = function(x,y,funcal);
            // printf("%f\n",funcal);
            // h_cal[i*n+j] = funcal;
            h_cal[i*n+j] = function(x,y);
            // printf("9\n");
            // printf("%f\t", h_cal[i*n+j]);
            y += dy;
        }
        // printf("\n");
        x += dx;
    }
    // printf("2\n");
    // std::cout<<std::endl<<std::endl;

    // std::cout<<'1'<<std::endl;

    // cudaMemcpy(x, h_x, n*(sizeof(double)), cudaMemcpyHostToDevice);
    // cudaMemcpy(y, h_y, n*(sizeof(double)), cudaMemcpyHostToDevice);
    hipMemcpy(sum, h_sum, n*n*(sizeof(double)), hipMemcpyHostToDevice);
    hipMemcpy(cal, h_cal, n*n*(sizeof(double)), hipMemcpyHostToDevice);

    // cal = function()

    saxpy<<<n,n>>>(n, dx, dy, cal, sum);

    hipMemcpy(h_sum, sum, n*n*(sizeof(double)), hipMemcpyDeviceToHost);

    double total = 0.;
    for(int i = 0; i<n; i++){
        for(int j = 0; j<n; j++){
            double k = h_sum[i+n*j];
            // printf("%f",k);
            total += k;
            // std::cout<<h_sum[i]<<std::endl;            
        }
        
    }

    *totalintegral += total;

    // printf("\n%f\t%f\n",total,*totalintegral);

    // cudaFree(x);
    hipFree(sum);
    hipFree(cal);

    // free(h_x);
    free(h_sum);
    free(h_cal);
}

int main(void)
{
    // printf("1");
    time_t start, end;

    start = time(NULL);

    // std::cout<<'1'<<std::endl;
    int n = 1000;

    // double a_1 = 0., b_1 = 10.;
    // double a_2 = 0., b_2 = 10.;

    double *totalintegral = (double*)malloc(sizeof(double));
    // double *totalintegral;
    
    // printf("1\n");
    // func(n,0.,10.,0.,10.,totalintegral);
    // printf("1\n");
    std::thread t1(func, n, 0., 5., 0., 5., totalintegral);
    std::thread t2(func, n, 0., 5., 5., 10., totalintegral);
    std::thread t3(func, n, 5., 10., 0., 5., totalintegral);
    std::thread t4(func, n, 5., 10., 5., 10., totalintegral);

    t1.join();
    t2.join();
    t3.join();
    t4.join();

    // printf("\n1\n");

    std::cout<<*totalintegral<<std::endl;

    end = time(NULL);
    std::cout<<"걸린 시간 : "<<double(end-start)<<std::endl;

    return 0;
}

#include <hip/hip_runtime.h>
#include <iostream>

// __global__ void function(float *x, int i, float dist){
//     dist = 3*x[i]*x[i];
// }

__device__ double function2(float x){
    double sqrSnn = 200.;
    double mp = 0.938272046;
    double m = 0.13957018;

    // double x = rapiditxnit(x);
    double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mpi
    // printf("2");
    double squareroot=sqrt(m*m+x*x);
    // printf("2");
    double xabs = fabs(x);
    double result = (squareroot/m)*exp(xabs-yb);

    return result;
}

__device__ double functions(float x){
    double y = function2(x)*x;
    double a = 0.5;
    double T = 0.5;
    double md = 1.;
    double m = 0.13957018;
    // return y;
    if(y>=1.){
        return 0.;
        // cal[index] = 0.;
    }
    else{
        return x*pow(1-y,a)*exp(-sqrt(m*m+x*x)/T)/sqrt(md*md+x*x);
        // cal[index] = x*pow(1-x,a)*exp(-sqrt(m*m+x*x)/T)/sqrt(md*md+x*x);
    }
}


//gpu를 이용하기 위해서는 '__global__'실행하려는 함수 앞에 붙여야 함.
__global__ void saxpy(int n, float a, float b, float *x, float dx, float *sum){
    // int i = blockIdx.x*blockDim.x;
    int i = blockIdx.x;
    // float dx = (b-a)/n;

    double fx = functions(x[i]);
    // printf("%f\n", fx);
    sum[i] = (fx)*dx;
    // sum[i] = exp(x[i])*dx;
    
    // float dist = function(x, i, dist);
    // sum[i] = dist*dx;

    // if( i < n )
    //     sum[i] = a*x[i]+b;
}

int main(void)
{
    // std::cout<<'1'<<std::endl;
    int n = 1000000;
    float dx;
    float total;
    // float *x, *sum;

    float *h_x = (float*)malloc(n*sizeof(float));
    float *h_sum = (float*)malloc(n*sizeof(float));

    // float x[10000] = {0.}, sum;

    float *x;
    float *sum;

    hipMalloc((void**) &x, n*sizeof(float));
    hipMalloc((void**) &sum, n*sizeof(float));

    float a = 0., b = 5.;

    // std::cout<<'1'<<std::endl;
    dx = (b-a)/float(n);
    for(int i = 0; i<n; i++){
        h_x[i] = a + ((b-a)/n)*i;
        // std::cout<<h_x[i]<<std::endl;
    }

    // std::cout<<'1'<<std::endl;

    hipMemcpy(x, h_x, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(sum, h_sum, n*sizeof(float), hipMemcpyHostToDevice);

    // std::cout<<'1'<<std::endl;

    saxpy<<<n,1>>>(n, a, b, x, dx, sum);

    // std::cout<<'1'<<std::endl;
    // 'saxpy'의 함수를 1xn개의 gpu thread가 실행한다는 의미
    // saxpy<<<b, n>>>();
    // 에서 saxpy는 함수의 이름, b는 함수를 수행할 block의 개수ㅏ n은 다시 하나의 thread block 안에 몇 개의 thread가 존재하는지를 정하는 것.

    hipMemcpy(h_sum, sum, n*sizeof(float), hipMemcpyDeviceToHost);

    total = 0.;
    for(int i = 0; i<n; i++){
        float k = h_sum[i];
        total += k;
    }

    hipFree(x);
    hipFree(sum);

    free(h_x);
    free(h_sum);

    std::cout<<total<<std::endl;

    return 0;
}
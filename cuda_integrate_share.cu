
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <thread>
#include <iomanip>


__device__ double lightcone(double pti, double yi, double sqrSnn, double mp, double m){

    double yb = acosh(sqrSnn/(2.*mp));
    double squareroot=sqrt(m*m+pti*pti);
    double result = (squareroot/m)*exp(fabs(yi)-yb);

    return result;

}

__device__ double integralAridge(double pti, double yi){
    // printf("%f\t%f\n", pti, yi);
    double sqrSnn = 200.;
    double mp = 0.938272046;
    double a = 0.5;
    double T = 0.5;
    double md = 1.;
    double m = 0.13957018;

    // printf("%f\t%f\n", pti, yi);
    // printf("1\n");
    double x = lightcone(pti, yi, sqrSnn, mp, m);
    // printf("%f, %f, %f\n", pti, yi, x);

    // double yi = rapidityinit(pti);
    // double yb = acosh(sqrSnn/(2.*mp));    //mN=mbeam, mb = mpi
    // double squareroot=sqrt(m*m+pti*pti);
    // double yiabs = std::fabs(yi);
    // // std::cout<<pti<<std::setw(8)<<yi<<std::setw(15)<<exp(yiabs-yb)<<std::setw(15)<<squareroot/mb<<std::endl;
    // double x = (squareroot/m)*exp(yiabs-yb);


    // double squareroot=sqrt(m*m+pti*pti);
    // printf("%f\t%f\n", pti, yi);
    if(x>=1.){
        return 0.;
        // cal[index] = 0.;
    }
    else{
        return pti*pow(1-x,a)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
        // cal[index] = pti*pow(1-x,a)*exp(-sqrt(m*m+pti*pti)/T)/sqrt(md*md+pti*pti);
    }
}

// cal의 x를 pti, y를 yi로 놓고 계산하자. -> x : block / y : thread
__global__ void integrate(int n, double pti_start, double pti_end, double *pti, double yi_start, double yi_end, double *yi, double *sum){
    // printf("%d\n",blockIdx.x);
    double dyi = ((yi_end-yi_start)/gridDim.x)*100;
    double dpti = (pti_end-pti_start)/blockDim.x;
    pti[blockIdx.x] = dpti*blockIdx.x+pti_start;
    yi[threadIdx.x] = dyi*threadIdx.x+yi_start;
    // printf("%d\n",blockIdx.x);
    // printf("%f\t%f\n", pti, yi);
    sum[blockIdx.x*blockDim.x+threadIdx.x] = integralAridge(pti[blockIdx.x], yi[threadIdx.x])*dyi*dpti;
    // printf("%d\n",blockIdx.x);
    // int index = blockIdx.x*blockDim.x+threadIdx.x;
    // cal[index] = integralAridge(cal[blockIdx.x], cal[blockIdx.y])
    // sum[index] = cal[index]*dx*dy;
}

// __global__ void integrate(double *pti, double dpti, double *yi, double dyi, double *sum){
//     // sum[blockIdx.x*gridDim.x+threadIdx.x] = integralAridge(pti[blockIdx.x], yi[threadIdx.x])*dyi*dpti;
//     sum[blockIdx.x+threadIdx.x*gridDim.x] = integralAridge(pti[blockIdx.x], yi[threadIdx.x])*dyi*dpti;
//     // printf("%f\t%f\n", pti[blockIdx.x], yi[threadIdx.x]);
// }

static const int blockSize = 1024;
__global__ void sumCommMultiBlock(double *gArr, int arraySize, double *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockDim.x;
    const int gridSize = blockDim.x*gridDim.x;
    double sum = 0;
    // printf("%f\n", gArr[threadIdx.x]);
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ double shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockDim.x/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

int main()
{
    clock_t start, finish;
    double duration;
    start = clock();
    using std::cout;
    using std::endl;
    using std::setw;

    // double dyi, dphii, sum, totalsum, phii, yi, dpti, pti, sum2, resultsum;
    double pti_start, pti_end, yi_start, yi_end, totalsum, Aridge;
    // double dyi, dpti;
    // int i, j, k, nyi, npti, nphii, check2;
    int nyi, npti;

    nyi = 100000;
    npti = 100000;

    pti_start = 0.;
    pti_end = 10.;
    yi_start = 0.;
    yi_end = 10.;

    // dyi = double((0.0+10.)/nyi);
    // dpti = double((0.+10.)/npti);

    // dyi = double ((yi_end-yi_start)/nyi);
    // dpti = double ((pti_end-pti_start)/npti);

    // dphii = double (M_PI+M_PI)/nphii;

    double *pti, *yi, *sum;
    // double *sum;

    // double *yi_cpu = (double*)malloc((nyi/100)*sizeof(double));
    // double *pti_cpu = (double*)malloc(npti*sizeof(double));
    double *sum_cpu = (double*)malloc(npti*(nyi/100)*sizeof(double));

    hipMalloc((void**) &pti, npti*sizeof(double));
    hipMalloc((void**) &yi, (nyi/100)*sizeof(double));
    hipMalloc((void**) &sum, npti*(nyi/100)*sizeof(double));

    // double sum_cpu = (double*)malloc(npti*nyi*sizeof(double));


    // sum = 0.;
    // pti = 0.0;  //적분을 pt=0부터 하는것이 옳은가? 원통좌표계에서의 적분인데?
    // yi = 0.;    //0~4 적분한 후 x2할 것.

    // for(int i = 0;i<npti;i++){
    //     pti_cpu[i] = ((10.-0.)/npti)*i+pti_start;
    // }
    // cudaMemcpy(pti,pti_cpu, npti*sizeof(double), cudaMemcpyHostToDevice);

    double* sumdist;
    hipMalloc((void**) &sumdist, npti*sizeof(double));

    totalsum = 0.;
    double totaldist = 0.;
    yi_end = 0.1;
    for(int k=0;k<100;k++){
        // for(int j=0;j<int(nyi/100);j++){
        //     yi_cpu[j] = ((10.-0.)/nyi)*j+yi_start;
        // }
        // cudaMemcpy(yi, yi_cpu, (nyi/100)*sizeof(double), cudaMemcpyHostToDevice);
        // cout<<"1"<<endl;
        // integrate<<<npti,int(nyi/100)>>>(pti, (pti_end-pti_start)/npti, yi, (yi_end-yi_start)/nyi, sum);
        integrate<<<npti,nyi/100>>>(k, pti_start, pti_end, pti, yi_start, yi_end, yi, sum);
        // cout<<"222222"<<endl;
        // cout<<"1"<<endl

        sumCommMultiBlock<<<npti,1024>>>(sum, npti*(nyi/100), sumdist);
        sumCommMultiBlock<<<1,1024>>>(sumdist, npti, sumdist);
        hipDeviceSynchronize();
        
        hipMemcpy(&totaldist, sumdist, sizeof(double), hipMemcpyDeviceToHost);
        printf("%f\n", totaldist);
        
        totalsum += totaldist;
        // printf("%f\t%f\t%f\n\n", yi_end, yi_start, totalsum);

        // cout<<"2"<<endl;
        yi_start += 0.1;
        yi_end += 0.1;
    }

    hipFree(pti);
    hipFree(yi);
    hipFree(sum);
    hipFree(sumdist);

    // free(pti_cpu);
    // free(yi_cpu);
    free(sum_cpu);

    totalsum *= 4.*M_PI;
    Aridge = 1/totalsum;
    cout<<totalsum<<setw(20)<<Aridge<<endl;
    finish = clock();
    duration = (double)(finish-start)/CLOCKS_PER_SEC;
    cout<<"time : "<<duration<<" sec"<<endl;

    return 0;
}




